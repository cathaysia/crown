#include <hip/hip_runtime.h>
#include <stdint.h>

__global__ void xor_bytes_kernel(uint8_t* inout, const uint8_t* in, uintptr_t size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx < size) {
        inout[idx] ^= in[idx];
    }
}

extern "C" hipError_t xor_bytes(uint8_t* inout, const uint8_t* in, uintptr_t size) {
    if(size == 0) {
        return hipError_t::hipSuccess;
    }

    hipGetErrorString(hipError_t::hipSuccess);

    int threadsPerBlock = 256;
    int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;

    xor_bytes_kernel<<<blocksPerGrid, threadsPerBlock>>>(inout, in, size);

    return hipGetLastError();
}
